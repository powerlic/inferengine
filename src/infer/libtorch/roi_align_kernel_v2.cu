#include "hip/hip_runtime.h"
// Modified from
// https://github.com/facebookresearch/detectron2/tree/master/detectron2/layers/csrc/ROIAlign
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved

#include "roi_align.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

template <typename T>
__device__ T bilinear_interpolate(const T* bottom_data, const int height,
                                  const int width, T y, T x,
                                  const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignForwardV2(
    const int nthreads, const T* bottom_data, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, T* top_data, bool aligned) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not use rounding; this implementation detail is critical
    T offset = aligned ? (T)0.5 : (T)0.0;
    T roi_start_w = offset_bottom_rois[1] * spatial_scale - offset;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale - offset;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale - offset;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale - offset;

    T roi_width = roi_end_w - roi_start_w;
    T roi_height = roi_end_h - roi_start_h;
    if (!aligned) {  // for backward-compatibility only
      roi_width = max(roi_width, (T)1.);
      roi_height = max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
                             ? sampling_ratio
                             : ceil(roi_height / pooled_height);  // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    // When the grid is empty, output zeros.
    const T count = max(roi_bin_grid_h * roi_bin_grid_w, 1);  // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++)  // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h +
                  static_cast<T>(iy + .5f) * bin_size_h /
                      static_cast<T>(roi_bin_grid_h);  // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
                    static_cast<T>(ix + .5f) * bin_size_w /
                        static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(offset_bottom_data, height, width, y, x,
                                     index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width, T y, T x, T& w1, T& w2, T& w3, T& w4,
    int& x_low, int& x_high, int& y_low, int& y_high,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

at::Tensor ROIAlignForwardV2Laucher(const torch::Tensor& input,
                                    const torch::Tensor& rois,
                                    const float spatial_scale,
                                    const int pooled_height,
                                    const int pooled_width,
                                    const int sampling_ratio, bool aligned) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");
  torch::TensorArg input_t{input, "input", 1}, rois_t{rois, "rois", 2};

  torch::CheckedFrom c = "ROIAlign_forward_cuda";
  torch::checkAllSameGPU(c, {input_t, rois_t});
  torch::checkAllSameType(c, {input_t, rois_t});
  at::cuda::CUDAGuard device_guard(input.device());

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty({num_rois, channels, pooled_height, pooled_width},
                          input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(at::cuda::ATenCeilDiv(static_cast<int64_t>(output_size), static_cast<int64_t>(512)), static_cast<int64_t>(4096)));
  dim3 block(512);

  if (output.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ROIAlign_forward", [&] {
    RoIAlignForwardV2<scalar_t><<<grid, block, 0, stream>>>(
        output_size, input.contiguous().data<scalar_t>(), spatial_scale,
        channels, height, width, pooled_height, pooled_width, sampling_ratio,
        rois.contiguous().data<scalar_t>(), output.data<scalar_t>(), aligned);
  });
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());
  return output;
}
